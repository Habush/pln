#include "hip/hip_runtime.h"
#include "AdviceData.h"
#include "DestinData.h"
#include "DestinKernel.h"

//pugiXML read/writer
#include "pugixml/pugixml.hpp"

#include <iostream>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <ctime>
#include <sstream>
#include <sys/stat.h>
#include <math.h>
// CUDA Lib
#include <hiprand.h>

#ifdef _WIN32
#include <direct.h>
#include <string>
#else
// Linux only requirements...
#include <errno.h>
#include <unistd.h>
#include <string.h>
#include <stdlib.h>
#endif

using namespace std;

// TODO: This main file is still pretty messy.
/*
 * It's almost a copy of the original main of DeSTIN where the custom file reader is replaced by a XML reader(pugixml)
 * Because of the quite basic understanding of the code and a more overall understanding of DeSTIN there are a lot of variables not been used.
 * Also cause there is still work in progress.
 * It still might be a idea to just replace the command line complete and do everything from the configuration
 * This will save a lot of code to analyze the input arguments
 */
void PrintHelp()
{
    // ***************************
    // Print out how to use DeSTIN
    // ***************************

    cout << "Usage: DestinCuda CodeWord MAXCNT LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    CodeWord must have 11 digits RRRRXXYYYYY" << endl;
    cout << "        RRRR  = 0000 to 9999 where 0000 is real random time." << endl;
    cout << "        XX    = 01 to 99 number of classes will not be higher then training file." << endl;
    cout << "        YYYYY = 00000 to 99999 number of examples of each class." << endl;
    cout << "                00000 means RANDOMLY PICK EXAMPLES until we finish clustering, period, up to max iterations." << endl;
    cout << "    MAXCNT is the number of digits we show it to train the unsupervised DeSTIN architecture" << endl;
    cout << "    LayerToShow = layer written to output file; it is given as S:E:O:P:T where " << endl;
    cout << "        S = first layer to write" << endl;
    cout << "        E = last layer to write" << endl;
    cout << "        O = offset for movements to write" << endl;
    cout << "        P = period of movements to write" << endl;
    cout << "        T = type.  Nothing (and no !) is beliefs.  Type can be: " << endl;
    cout << "            A is belief in advice states computed by tabular method." << endl;
    cout << "            N is belief in advice states computed by neural network function approximator." << endl;
    cout << "            L is belief in advice states computed by linear function approximator." << endl;
    cout << "    ParamsFile is a file that has the run parameters" << endl;
    cout << "    TrainingDataFile is the binary data file for training.  A testing file with the SAME NAME and appended with _TESTING is assumed" << endl;
    cout << "    DestinOutputFile is the name of the DeSTIN network output file for saving." << endl;
    cout << "         Use -D as default, which is the experiment number with a .xml at the end, in the TargetDirectory directory" << endl;
    cout << "    TargetDirectory is where we want to put the MAIN OUTPUT DATA FILES.  We ALWAYS write an experiment marker to the " << endl;
    cout << "        ../DiagnosticData area.  But if you are writing out a lot of data you can specify another directory." << endl;
    cout << "        Put D for default which is the ../DiagnosticData area." << endl;
    cout << "    [OutputDistillationLevel] is optional.  If this exists it must be a number and currently its got to be 0.  "<<endl;
    cout << "        0 = regular outputs with a lot of details about movements and processing: this is our input to SampleAndStack"<<endl;
    cout << "        1 = outputs compatible with the regular distilled output of SampleAndStack. If you use this you can skip SampleAndStack.exe" << endl;
    cout << endl;
    cout << "-OR-" << endl;
    cout << endl;
    cout << "Usage: DestinCuda -F InputNetworkFile LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    -F signifies use a saved DeSTIN network file " << endl;
    cout << "    InputNetworkFile is the NAME of the saved DeSTIN network file" << endl;
    cout << "    All others are as in first usage type" << endl;
    cout << endl;
}

bool FileExists(string strFilename)
{
    // **************************
    // Does the given file exists
    // **************************
    // For detailed information look the return values of stat

    struct stat stFileInfo;
    bool blnReturn;
    int intStat;

    // Attempt to get the file attributes
    intStat = stat(strFilename.c_str(),&stFileInfo);
    if(intStat == 0) {
        // File exists
        blnReturn = true;
    }
    else
    {
        // File not exists or no permission
        blnReturn = false;
    }

    return(blnReturn);
}

string GetNextFileForDiagnostic()
{
    // *************************************
    // Find next available experimental file
    // *************************************
    // Check if there is a previous experiment inside ../DiagnosticData

    string strFileName;
    int iExperimentNumber=-1;
    bool bFileFound = true;
    while ( bFileFound )
    {
        iExperimentNumber++;
        stringstream buffer;
        buffer << "../DiagnosticData/DestinDiagnostics" << iExperimentNumber << "-0.xml";
        strFileName =  buffer.str();

        bFileFound = FileExists(strFileName);
        strFileName.erase(strFileName.length()-6,2);
    }
    strFileName = strFileName.substr(18);

    return strFileName;
}

void GetParameters( const char* cFilename, int& NumberOfLayers, double*& dcMu, double*& dcSigma, double*& dcRho,
                    int*& NumberOfStates, bool& bAveraging,bool& bFFT,bool& bBinaryPOS,int* DistanceMeasureArray,
                    bool& bUseStarvationTrace,int& PSSAUpdateDelay,bool& bIgnoreAdvice,
                    int**& SEQ, int& SEQ_LENGTH, string& sFileContents, int& iBlocksToProcess,
                    bool& bBasicOnlineClustering,
                    bool& bClanDestin, bool& bInitialLayerIsTransformOnly,bool& bUseGoodPOSMethod,
                    int*& RowsPerLayer, float*& FixedLearningRateLayer, bool*& bSelfAndUpperFeedback, int& LastLayerInputX, int& LastLayerInputY )
{
    // ******************************************
    // Read the XML config file (parameters file)
    // ******************************************
    // This function is rewritten and is not backwards compatible with the DestinPort one.
    // Instead of a txt file its now a XML file pugixml is used for parsing it.
    ifstream stmInput(cFilename);
    string sBuffer;
    // Put the config file into a vector and as one big string back to sFileCOntents
    while ( getline(stmInput, sBuffer) )
    {
        sFileContents = sFileContents + "~" + sBuffer + "\n";
    }
    stmInput.close();

    pugi::xml_document xFile;

    pugi::xml_parse_result result = xFile.load_file(cFilename);
    std::cout << "XML config file Load result: " << result.description() << endl;
    if ( result )
    {
        // Root node is destin
        pugi::xml_node root = xFile.child("destin");

        // Retrieve SEQ_LENGTH
        pugi::xml_node seq = root.child("seq");
        SEQ_LENGTH = seq.attribute("length").as_int();

        // Retrieve all steps
        SEQ = new int*[SEQ_LENGTH];
        pugi::xml_node step = seq.child("step");
        for( int iStep = 0; iStep < SEQ_LENGTH; iStep++ )
        {
            SEQ[iStep]=new int[2];
            SEQ[iStep][0] = step.attribute("x").as_int();
            SEQ[iStep][1] = step.attribute("y").as_int();
            step = step.next_sibling("step");
        }

        // Retrieve amount of layers
        pugi::xml_node layers = root.child("layers");
        NumberOfLayers = layers.attribute("value").as_int();
        LastLayerInputX = layers.attribute("inputX").as_int();
        LastLayerInputY = layers.attribute("inputY").as_int();

        // Retrieve configuration each layer
        dcMu = new double[NumberOfLayers];
        dcSigma = new double[NumberOfLayers];
        dcRho = new double[NumberOfLayers];
        NumberOfStates = new int[NumberOfLayers];
        DistanceMeasureArray = new int[NumberOfLayers];
        RowsPerLayer = new int[NumberOfLayers];
        FixedLearningRateLayer = new float[NumberOfLayers];
        bSelfAndUpperFeedback = new bool[NumberOfLayers];
        pugi::xml_node layer = layers.child("layer");
        // Loop true each layer configuration
        for( int iLayer = 0; iLayer < NumberOfLayers; iLayer++ )
        {
            dcMu[iLayer] = layer.attribute("mu").as_double();
            dcSigma[iLayer] = layer.attribute("sigma").as_double();
            dcRho[iLayer] = layer.attribute("rho").as_double();
            NumberOfStates[iLayer] = layer.attribute("states").as_int();
            DistanceMeasureArray[iLayer] = layer.attribute("distance").as_int();
            RowsPerLayer[iLayer] = layer.attribute("rowsColmsPerLayer").as_int();
            FixedLearningRateLayer[iLayer] = layer.attribute("fixedLearningRate").as_float();
            bSelfAndUpperFeedback[iLayer] = layer.attribute("selfAndUpperFeedback").as_bool();
            layer = layer.next_sibling("layer");
        }

        // Retrieve settings for overal DeSTIN
        pugi::xml_node settings = root.child("settings");
        bAveraging = settings.child("averaging").attribute("value").as_bool();
        bFFT = settings.child("fft").attribute("value").as_bool();
        bBinaryPOS = settings.child("binaryPos").attribute("value").as_bool();
        bUseStarvationTrace = settings.child("starvationTrace").attribute("value").as_bool();
        PSSAUpdateDelay = settings.child("pssaDelay").attribute("value").as_int();
        bIgnoreAdvice = settings.child("ignoreAdvice").attribute("value").as_bool();
        iBlocksToProcess = settings.child("processingBlockSize").attribute("value").as_int();
        bBasicOnlineClustering = settings.child("basicOnlineClustering").attribute("value").as_bool();
        bClanDestin = settings.child("clanDestin").attribute("value").as_bool();
        bInitialLayerIsTransformOnly = settings.child("initialLayerIsTransformOnly").attribute("value").as_bool();
        bUseGoodPOSMethod = settings.child("useGoodPOSMethod").attribute("value").as_bool();
    }
    else
    {
        std::cout << "Error description: " << result.description() << "\n";
        std::cout << "Error offset: " << result.offset << " (error at [..." << (cFilename + result.offset) << "]\n\n";
    }
    cout << "------------------" << endl;
}

bool CreateDestinOnTheFly(string ParametersFileName, int& NumberOfLayers, DestinKernel*& DKernel,
                          DestinData& DataSourceForTraining, int& SEQ_LENGTH, int**& SEQ,
                          int*& ImageInput)

{
    // *********************
    // Create DeSTIN network
    // *********************

    double* dcMu;
    double* dcSigma;
    double* dcRho;
    int* NumberOfCentroids;
    bool bAveraging;
    bool bFFT;
    bool bBinaryPOS;
    int DistanceMeasureArray[128];
    bool bUseStarvationTrace;
    int PSSAUpdateDelay;
    bool bIgnoreAdvice;
    string sParametersFileContents;
    int iBlocksToProcess;
    bool bBasicOnlineClustering;
    bool bClanDestin;
    bool bInitialLayerIsTransformOnly;
    bool bDoGoodPOS;
    int* RowsPerLayer;
    float* FixedLearningRateLayer;
    bool* bSelfAndUpperFeedback;
    ImageInput = new int[2];

    GetParameters( ParametersFileName.c_str(), NumberOfLayers, dcMu, dcSigma, dcRho, NumberOfCentroids,
                   bAveraging, bFFT, bBinaryPOS, DistanceMeasureArray,
                   bUseStarvationTrace, PSSAUpdateDelay, bIgnoreAdvice, SEQ, SEQ_LENGTH,
                   sParametersFileContents, iBlocksToProcess,
                   bBasicOnlineClustering, bClanDestin, bInitialLayerIsTransformOnly, bDoGoodPOS,
                   RowsPerLayer, FixedLearningRateLayer, bSelfAndUpperFeedback, ImageInput[0], ImageInput[1]);

    // The name and loop looks like it is giving to option to save steps of the movements.
    vector<bool> vectorOfMovementsToSave;
    for( int c=0;c<SEQ_LENGTH;c++ )
    {
        vectorOfMovementsToSave.push_back(false);
    }

    DKernel = new DestinKernel[NumberOfLayers];
    int* ColsPerLayer = new int[NumberOfLayers];
    int* NumberOfParentStates = new int[NumberOfLayers];
    int* InputDimensionality = new int[NumberOfLayers];
    int* OffsetSelf = new int[NumberOfLayers];
    int* OffsetSelfFeedback = new int[NumberOfLayers];

    int MovementsForClusteringOption = 1;
    if ( bClanDestin )
    {
        MovementsForClusteringOption=SEQ_LENGTH; // use this if you want one clustering engine per movement
    }

    // These are changed inside the XML file
    if ( bFFT )
    {
        InputDimensionality[0] = 10;  //4x4 FFT has 10 unique magnitude values...
    }
    else
    {
        InputDimensionality[0] = 16;  //4x4 has 16 inputs
    }

    if ( bInitialLayerIsTransformOnly )
    {
        InputDimensionality[1] = NumberOfCentroids[0];
        NumberOfParentStates[0] = NumberOfCentroids[1];
        RowsPerLayer[1] = RowsPerLayer[0];
        for( int Layer=2; Layer<NumberOfLayers; Layer++ )
        {
            InputDimensionality[Layer] = 4*NumberOfCentroids[Layer-1];
            NumberOfParentStates[Layer-1] = NumberOfCentroids[Layer];
            RowsPerLayer[Layer] = RowsPerLayer[Layer-1]/2;
        }
    }
    else
    {
        for(int Layer=1; Layer<NumberOfLayers; Layer++ )
        {
            InputDimensionality[Layer] = 4*NumberOfCentroids[Layer-1];
            NumberOfParentStates[Layer-1] = NumberOfCentroids[Layer];
        }
    }
    NumberOfParentStates[NumberOfLayers-1]=1;
    //if you want to FORCE stability, an exponential / gaussian decay will start at iDecayPoint
    bool bUseDecayLR = false;
    int DigitToStartDecay=1000;
    int iDecayPoint = SEQ_LENGTH*DigitToStartDecay;
    float fRhoThreshold = (float)(1e-2);
    bool bUseRhoDerivative = false;
    int MaxNumberOfInputs=-1;
    int MaxNumberOfOutputs=-1;

    // hiprandGenerator_t is a CUDA version of rand
    // This fills the whole memory block with number between 0.0 and 1.0
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // TODO: Add seed code instead of 1
    // This is the right place to do this saves the most time creating numbers. (Inside layer increase the time by +/- 5 times)
    hiprandSetPseudoRandomGeneratorSeed( gen, 1 );

    // Here we put the first image into the device memory
    for( int Layer=0; Layer<NumberOfLayers; Layer++ )
    {
        bool bTopNode = false;
        bool bAveragingLayer = false;
        bool bConstrainInitialCentroids = true;

        OffsetSelf[Layer] = InputDimensionality[Layer]; // the basic value.
        ColsPerLayer[Layer] = RowsPerLayer[Layer];
        // increase the input dimensionality if you are using the self-upper feedback
        if ( bSelfAndUpperFeedback[Layer] )
        {
            InputDimensionality[Layer] = InputDimensionality[Layer]+NumberOfCentroids[Layer];  // layers get self-feedback
            OffsetSelfFeedback[Layer] = InputDimensionality[Layer];
            if ( Layer<NumberOfLayers-1 )
            {
                InputDimensionality[Layer] = InputDimensionality[Layer]+NumberOfCentroids[Layer+1];  // all layers but the top get feedback from above
            }
        }
        // Initial layer does this a little different (input is raw instead of centroids)
        if ( Layer==0 )
        {
            bAveragingLayer=bAveraging;
            bConstrainInitialCentroids=false;
        }
        if ( Layer==NumberOfLayers-1)
        {
            // Yes you are the top layer
            bTopNode = true;
        }

        DKernel[Layer].Create( Layer, RowsPerLayer[Layer], ColsPerLayer[Layer], NumberOfCentroids[Layer], InputDimensionality[Layer], FixedLearningRateLayer[Layer], gen);
        // Assign Childeren and Parrents of nodes
        if ( NumberOfCentroids[Layer] > MaxNumberOfOutputs )
        {
            MaxNumberOfOutputs=NumberOfCentroids[Layer];
        }
        if ( InputDimensionality[Layer] > MaxNumberOfInputs )
        {
            MaxNumberOfInputs=InputDimensionality[Layer];
        }
    }
    // The generator have to be destroyed after use.
    hiprandDestroyGenerator( gen );
    cout << "------------------" << endl;
    return 0;
}

struct CommandArgsStuc {

	bool bCreateFromFile;
        string strDestinNetworkFileToRead;
};

/**
 *  parseCommandArgs
 *
 *  Parses the program command line arguments and fills
 *  the out CommandArgsStuc appropriately ( which you can
 *  see right now only consists of bCreateFromFile ).
 *
 *  This almost doesn'thave any side affects at the moment
 *  because most of the declared variables in this function
 *  are not used because most of the functionality it represents
 *  is not yet  ported from the single cpu version of Destin.
 */
int parseCommandArgs(  string strDiagnosticFileName, int argc, char* argv[], CommandArgsStuc &out){
    // Argument: TargetDirectory
    // A given location instead or default
    string strDiagnosticDirectoryForData;
    string strArg7 = argv[7];
    if ( strArg7 == "D" )
    {
        strDiagnosticDirectoryForData = "../DiagnosticData/";
    }
    else
    {
        // Buffer with path + filename where to put diagnostic data
        stringstream buffer;
        buffer << strArg7.c_str() << "/";
        strDiagnosticDirectoryForData = buffer.str();
    }

    // Argument: DestinOutputFile or InputNetworkFile

    string strDestinNetworkFileToWrite;
    string FirstArg = argv[1];
    if ( FirstArg=="-F" )
    {
        // Argument: InputNetworkFile
        out.bCreateFromFile = true;
        out.strDestinNetworkFileToRead = argv[2];  // we read from this file...

        if ( !FileExists( out.strDestinNetworkFileToRead ) )
        {
            cout << "designated input network file named " << out.strDestinNetworkFileToRead.c_str() << " does not exist" << endl;
            return 0;
        }
        cout << "Writing destin file to: " << strDestinNetworkFileToWrite << endl;
    }
    else
    {
        // Argument: DestinOutputFile
        out.bCreateFromFile = false;
        strDestinNetworkFileToWrite = argv[6]; // we write to this file, and then we read from it too!!
        if ( strDestinNetworkFileToWrite == "-D" )
        {
            // If given -D
            strDestinNetworkFileToWrite= strDiagnosticDirectoryForData + strDiagnosticFileName;
            cout << "Writing default destin file to: " << strDestinNetworkFileToWrite << endl;
        }
        out.strDestinNetworkFileToRead = strDestinNetworkFileToWrite;
    }


    // Create the old variable sDiagnosticFileNameForMarking
    // it have the location based on TargetDirectory + FileName
    string sDiagnosticFileNameForMarking;
    sDiagnosticFileNameForMarking = strDiagnosticDirectoryForData + strDiagnosticFileName;
    // Argument: LayerToShow
    // Structure of processing S:E:O:P:T
    // List of default values
    int FirstLayerToShowHECK = 3;
    int LastLayerToShow = FirstLayerToShowHECK;
    int iMovementOutputOffset = 0;
    int iMovementOutputPeriod = 1;
    OutputTypes eTypeOfOutput = eBeliefs;

    string sLayerSpecs = argv[3];
    int iColon = sLayerSpecs.find(":");
    if ( iColon == -1 || sLayerSpecs.substr(iColon).empty() )  //first layer = last layer, and no sampling specified.
    {
        // S
        FirstLayerToShowHECK=atoi(sLayerSpecs.c_str());
        LastLayerToShow=FirstLayerToShowHECK;
    }
    else
    {
        // S:E
        FirstLayerToShowHECK=atoi(sLayerSpecs.substr(0,1).c_str());
        LastLayerToShow=atoi(sLayerSpecs.substr(iColon+1,1).c_str());
        sLayerSpecs = sLayerSpecs.substr(iColon+1);
        iColon = sLayerSpecs.find(":");
        if ( iColon!=-1 && !( sLayerSpecs.substr(iColon).empty() ) )
        {
            //S:E:O
            sLayerSpecs = sLayerSpecs.substr(iColon+1);
            iMovementOutputOffset = atoi(sLayerSpecs.substr(0,1).c_str());
            iColon = sLayerSpecs.find(":");
            if ( iColon!=-1 && !( sLayerSpecs.substr(iColon).empty() ) )
            {
                //S:E:O:P
                sLayerSpecs = sLayerSpecs.substr(iColon+1);
                iMovementOutputPeriod = atoi(sLayerSpecs.substr(0,1).c_str());
                iColon = sLayerSpecs.find(":");
                if ( iColon!=-1 && !( sLayerSpecs.substr(iColon).empty() ) )
                {
                    //S:E:O:P:T
                    sLayerSpecs = sLayerSpecs.substr(iColon+1);
                    if ( sLayerSpecs.substr(0,1)=="A" )
                    {
                        eTypeOfOutput = eBeliefInAdviceTabular;
                    }
                    else if ( sLayerSpecs.substr(0,1)=="B" )
                    {
                        eTypeOfOutput = eBeliefs;
                    }
                    else if ( sLayerSpecs.substr(0,1)=="N" )
                    {
                        eTypeOfOutput = eBeliefInAdviceNNFA;
                    }
                    else if ( sLayerSpecs.substr(0,1)=="L" )
                    {
                        eTypeOfOutput = eBeliefInAdviceLinearFA;
                    }
                    else
                    {
                        cout << "Do not understand the output type " << sLayerSpecs.c_str() << endl;
                        return 1;
                    }
                }
            }
        }
    }

    // Optional argument: OutputDistillationLevel
    // This will write out a distilled movement log file this movement log matches that what SampleAndStack would produce.
    int OutputDistillationLevel = 0; //default level
    if ( argc == 9 )
    {
        OutputDistillationLevel = atoi(argv[8]);
    }


    return 0;
}


int MainDestinExperiments(int argc, char* argv[])
{
    time_t destinStart = time(NULL);
    // ********************************************
    // Main experiment of DeSTIN (Also called main)
    // ********************************************

    // File for diagnostic
    string strDiagnosticFileName;
    strDiagnosticFileName = GetNextFileForDiagnostic();

    // arguments processing

    // For debug information we output the command line to our Diagnostic file.
    string strCommandLineData = "";
    for( int i=0; i<argc; i++ )
    {
        strCommandLineData += argv[i];
        strCommandLineData += " ";
    }


    CommandArgsStuc argsStruc;
    if(parseCommandArgs(strDiagnosticFileName, argc, argv, argsStruc)!=0){
    	return 1;
    }


    // **********************
    // Loading data source(s)
    // **********************
    // Arguments: TrainingDataFile
    // Load the training file for DeSTIN
    string strDestinTrainingFileName = argv[5];

    // Data object containing source (training)
    DestinData DataSourceForTraining;

    int NumberOfUniqueLabels;
    DataSourceForTraining.LoadFile(strDestinTrainingFileName.c_str());
    NumberOfUniqueLabels = DataSourceForTraining.GetNumberOfUniqueLabels();
    if ( NumberOfUniqueLabels==0 )
    {
        cout << "There seems to be something off with data source " << strDestinTrainingFileName.c_str() << endl;
        return 0;
    }

    // A vector with all the labels of the data source
    vector<int> vLabelList;
    DataSourceForTraining.GetUniqueLabels(vLabelList);

    // Load the test file for DeSTIN
    string strTesting = strDestinTrainingFileName;
    strTesting = strTesting + "_TESTING";
    // Data object of test source
    DestinData DataSourceForTesting;

    DataSourceForTesting.LoadFile((char*)(strTesting.c_str()));
    if ( DataSourceForTesting.GetNumberOfUniqueLabels()!=NumberOfUniqueLabels )
    {
        cout << "Test set does not have the same number of labels as train set " << endl;
        return 0;
    }

    // **************************
    // Preparing working data set
    // **************************
    // now get the data set creation parameters
    int NumberOfUniqueLabelsToUse;
    int MAX_CNT = 1000;
    int iTestSequence = 0;
    string ParametersFileName;
    vector< pair<int,int> > vIndicesAndGTLabelToUse;

    if (argsStruc.bCreateFromFile==false )
    {
        // Argument: MAXCNT
        MAX_CNT=atoi(argv[2]);
        // Argument: CodeWord
        iTestSequence=atoi(argv[1]);
        string sCodeWord=argv[1];
        if (sCodeWord.length() != 11 )
        {
            PrintHelp();
            return 0;
        }
        // First part of code word RRRR = for time seeding
        string sNumInp;
        sNumInp= sCodeWord.substr(0,4);

        // if the first 4 digits are 0000 make a TRUE random, otherwise use the complete number.
        int iReserve = atoi( sNumInp.c_str() );
        if ( iReserve == 0 )
        {
            srand( time(NULL) );
        }
        else
        {
            int iRandSeed = iTestSequence;
            srand( (unsigned int)iRandSeed );
        }

        // Second part of code word XX = number of inputs
        sNumInp = sCodeWord.substr(4,2);
        NumberOfUniqueLabelsToUse = atoi( sNumInp.c_str() );

        // Last part of code word YYYYY
        int iNumberOfExamplesFromEachLabel;
        sNumInp = sCodeWord.substr(6,5);
        iNumberOfExamplesFromEachLabel=atoi( sNumInp.c_str() );

        // if iNumberOfExamplesFromEachLabel is 0 we randomly pick examples from the available
        // classes and only show them ONE TIME
        // Generate the examples from the dictates given here.
        vector< pair<int,int> > LabelsAndIndicesForUse;
        cout << "------------------" << endl;
        int DestinTrainSampleStep = 1;
        if(iNumberOfExamplesFromEachLabel == 0)
        {
            DestinTrainSampleStep = 25;
        }
        for(int iLabel=0;iLabel<NumberOfUniqueLabelsToUse;iLabel++)
        {
            int cnt = 0;
            vector<int> IndicesForThisLabel;
            DataSourceForTraining.GetIndicesForThisLabel(iLabel,IndicesForThisLabel);
            if ( IndicesForThisLabel.size() > iNumberOfExamplesFromEachLabel && iNumberOfExamplesFromEachLabel != 0)
            {
                for(int jj=0;jj<iNumberOfExamplesFromEachLabel;jj++)
                {
                    cnt++;
                    pair<int,int> P;
                    P.first = IndicesForThisLabel[jj];
                    P.second = iLabel;
                    LabelsAndIndicesForUse.push_back(P);
                }
            }
            else
            {
                for(int jj=0;jj<IndicesForThisLabel.size();jj=jj+DestinTrainSampleStep)
                {
                    cnt++;
                    pair<int,int> P;
                    P.first = IndicesForThisLabel[jj];
                    P.second = iLabel;
                    LabelsAndIndicesForUse.push_back(P);
                }

            }
            cout << "Label: " << iLabel << " got " << cnt << " unique sample(s)." << endl;
        }
        iNumberOfExamplesFromEachLabel = LabelsAndIndicesForUse.size()/NumberOfUniqueLabelsToUse;

        // Now generate MAX_CNT+1000 random numbers from 0 to LabelsAndIndicesForUse-1
        // and use these to populate vIndicesAndGTLabelToUse

        // Debug list of labels to be used
        int * Picked;
        Picked = (int *) malloc(sizeof(int) * NumberOfUniqueLabels);

        for(int jj=0;jj<NumberOfUniqueLabels;jj++)
        {
            Picked[jj]=0;
        }

        int Digit;
        int iChoice;
        for(int jj=0;jj<MAX_CNT;jj++)
        {
            //pick the digit first...
            Digit = rand() % NumberOfUniqueLabelsToUse;
            iChoice = Digit * iNumberOfExamplesFromEachLabel;
            iChoice = iChoice+rand() % iNumberOfExamplesFromEachLabel;

            pair<int,int> P;
            P = LabelsAndIndicesForUse[iChoice];

            vIndicesAndGTLabelToUse.push_back( P );
            // Debug counter of labels used by label
            Picked[P.second] += 1;
        }

        // Debug information on amount of examples we use each label
        cout << "------------------" << endl;
        for(int jj=0;jj<NumberOfUniqueLabels;jj++)
        {
            cout << "Label: " << jj << " will show " << Picked[jj] << " sample(s)." << endl;
        }
        free( Picked);
        cout << "------------------" << endl;
    }  //check on bCreateFromFile==false
    else
    {
        // TODO: We want to create the network from an INPUT FILE!
        cout << "We want to create the network from an INPUT FILE!" << endl;
    }

    // Argument: ParamsFile
    // A configuration file for DeSTIN
    ParametersFileName=argv[4];
    if ( !FileExists(ParametersFileName) )
    {
        // According to the help the ParamsFile is always used? Maybe some vital information on how to load data?
        // Or some testing to see how the network reacts when expanding or shrinking the network.
        cout << "Parameters file name does not exist" << endl;
        return 0;
    }

    // ***********************
    // Creating DeSTIN network
    // ***********************
    // Yes its going to happen we going to create the network where we are waiting for.
    int SEQ_LENGTH = 0;
    int** SEQ;
    int* ImageInput;

    DestinKernel* DKernel=NULL;
    map<int,int> LabelsUsedToCreateNetwork;
    map<int,int> IndicesUsedToCreateNetwork;
    int NumberOfLayers=4;
    if ( !argsStruc.bCreateFromFile)
    {
        int LayerToShow=-1;   //normally this should be -1 for regular operation.  For debugging, set it to 0 to look at the particular input for layer 0
        int RowToShowInputs=3;
        int ColToShowInputs=3;
        CreateDestinOnTheFly(ParametersFileName, NumberOfLayers, DKernel,
                             DataSourceForTraining, SEQ_LENGTH, SEQ, ImageInput);

        for (int i=0; i<NumberOfLayers;i++)
        {
            cout << "DeSTIN Layer information" << endl;
            cout << "Layer: " << DKernel[i].GetID() << endl;
            cout << "Dimension (row, col): " << DKernel[i].GetNumberOfRows() << " X " << DKernel[i].GetNumberOfCols() << endl;
            cout << "Input each node: " << DKernel[i].GetNumberOfInputDimensionlity() << endl;
            cout << "Centroids: " << DKernel[i].GetNumberOfStates() << endl;
            cout << endl;
        }
    }
    else
    {
        // even if you don't create the file here, we want to mark the experiment number so make a dummy file...
        ofstream stmDummy;
        stmDummy.open(strDiagnosticFileName.c_str(),ios::out);
        stmDummy << strCommandLineData.c_str() << endl;
        stmDummy << "DummyHeader" << endl;
        stmDummy.close();
    }

    cout << "------------------" << endl;
    cout << "Run Destin" << endl;
    cout << "Images to be processed: " << MAX_CNT << endl;
    cout << "Each image moves: " << SEQ_LENGTH << " times." << endl;

    double procces = 0.1;
    for(int i=0;i<MAX_CNT;i++)
    {
        if(i > (MAX_CNT-1)*procces)
        {
            cout << procces*100 << "%" << endl;
            procces+=0.1;
        }
        stringstream xml;
        xml << "<destin>" << endl;

        pair<int,int> element = vIndicesAndGTLabelToUse[i];
        int indexOfExample = element.first;
        int label = element.second;
        time_t iStart = time(NULL);
        for(int seq=0;seq<SEQ_LENGTH;seq++)
        {
            stringstream xmlLayer;
            // Run lowest layer (Kernel)

            time_t lStart = time(NULL);
            DataSourceForTraining.SetShiftedDeviceImage(indexOfExample, SEQ[seq][0], SEQ[seq][1], ImageInput[0], ImageInput[1]);
            DKernel[0].DoDestin(DataSourceForTraining.GetPointerDeviceImage(),xmlLayer);
            for(int i=1;i<NumberOfLayers;i++)
            {
                DKernel[i].DoDestin(DKernel[i-1].GetDevicePointerOutput(),xmlLayer);
            }
            time_t lStop = time(NULL);
            xmlLayer << "<layerRuntime>" << lStop-lStart << "</layerRuntime>" << endl;
            if(seq == SEQ_LENGTH-1)
            {
                xml << xmlLayer.str().c_str();
            }
            xmlLayer.clear();
        }
        time_t iStop = time(NULL);
        xml << "<image id=\"" << i << "\" label=\"" << label << "\" labelIndex=\"" << indexOfExample << "\" runtime=\"" << iStop-iStart << "\" />" << endl;
        xml << "</destin>" << endl;
        if(i == MAX_CNT-1)
        {
            pugi::xml_document outputFile;
            outputFile.load(xml.str().c_str());
            string file = argsStruc.strDestinNetworkFileToRead;
            stringstream num;
            num << "-" << i;
            file.insert(file.length()-4, num.str());
            outputFile.save_file(file.c_str());
        }
    }
    time_t destinStop = time(NULL);
    cout << "Time run: " << destinStop-destinStart << endl;

    free(DKernel);

    return 0;
}

// Simple run command: destinCuda(.exe) 00010100000 120 2:3 ./config.xml ../../data/MNISTTraining32 -D D
int main(int argc, char* argv[])
{
    // ********************
    // Startup check DeSTIN
    // ********************
    // There should be 8 or 9 arguments at this time if not show how to use DeSTIN
    if ( argc==8 || argc==9 )
    {
        cout << "Starting DeSTIN" << endl;
        cout << "------------------" << endl;
        return MainDestinExperiments(argc,argv);
    }
    else
    {
        PrintHelp();
        return 0;
    }
}
