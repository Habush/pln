#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>

// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int AmountThreads = 128;

using namespace std;

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, float *CentroidStarvation);
__global__ void CalculateWinningCentroids( int States, float *CentroidDist, int *WinningCentroids );
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation );
__global__ void UpdateWinningCentroids( int States, int InputDimensionlity, float LearningRate, float *InputData, float *CentroidVectorData, int *WinningCentroids, float *CentroidDist );
__global__ void CalculatePOS( int States, float *CentroidDist, float *Output );

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
    mLearningRate = 0;
    mSTARVATION_COEFFICIENT = 0;
	hipGetDeviceCount(&mDevices);
	cout << "Kernel created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    hipFree( dCentroidsVectorData );
    hipFree( dNodeOutput );
    hipFree( dCentroidsDistance );
    hipFree( dCentroidStarvation );
    hipFree( dWinningCentroids );
    free ( mCentroidsDistance );
    free ( mCentroidStarvation );
    free ( mWinningCentroids );
    free ( mNodeOutput );
    free(mCentroidWinCounter);
    cout << "Kernel destroyed" << endl;
}

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int InputDimensionlity, float FixedLeaningRate, hiprandGenerator_t gen)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;
    mLearningRate = FixedLeaningRate;

    mSTARVATION_COEFFICIENT = 1.0/((float)InputDimensionlity*(float)InputDimensionlity);
    if ( mSTARVATION_COEFFICIENT < 1.0/512.0 )
    {
        mSTARVATION_COEFFICIENT=1.0/512.0;
    }

    // Define the data sizes
    // Size of de nodes is rows times columns
    sizeOfNodes = mRows*mCols;
    // Size of the data of nodes is rows times columns times centroids
    sizeOfNodeData = sizeOfNodes*mStates;

    // Size of the layer with all vectors is rows times columns times centroids times the input (also observation) vector length.
    sizeOfLayerData = sizeOfNodeData*mInputDimensionlity;
    // Keep track of which centroid won
    mCentroidWinCounter = new int[sizeOfNodeData];
    for(int c=0;c<sizeOfNodeData;c++)
    {
        mCentroidWinCounter[c] = 0;
    }

    // Array full with all the winning centroids of each node
    mWinningCentroids = new int[sizeOfNodes];
    hipMalloc( (void**)&dWinningCentroids, sizeOfNodes*sizeof(int) );

    // Node data contains the distance to the observation of all centroids (It's is empty the first run)
    mCentroidsDistance = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidsDistance, sizeOfNodeData*sizeof(float) );

    // Starvation data for all centroids
    mCentroidStarvation = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidStarvation, sizeOfNodeData*sizeof(float) );
    for(int i=0;i<sizeOfNodeData;i++)
    {
        mCentroidStarvation[i]=1.0f;
    }
    // Copy the data from host to device
    hipMemcpy(dCentroidStarvation, mCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyHostToDevice);

    // Output for next layer
    mNodeOutput = new float[sizeOfNodeData];
    hipMalloc( (void**)&dNodeOutput, sizeOfNodeData*sizeof(float) );

    hipMalloc( (void**)&dCentroidsVectorData, sizeOfLayerData*sizeof(float) );
    // This is to fill the dLayerData with all random numbers between 0.0 and 1.0
    hiprandGenerateUniform( gen, dCentroidsVectorData, sizeOfLayerData );
}

void DestinKernel::DoDestin( float *Input, stringstream& xml )
{
    // Threads is the amount of thread inside each block
    dim3 threads( AmountThreads );
    // Grid is the amount of blocks inside a grid.
    dim3 grid( mCols, mRows );
    // Cause of the use of dynamic shared memory you have to tell the kernel how much shared memory space you need for each block.
    int sharedMem;
    // The launch of the kernels itself with centroids(states), dimension, input data and the Data of the layer itself
    // Calculating the distance of the centroids to an observation
    sharedMem = (mInputDimensionlity+mInputDimensionlity)*sizeof(float);
    CalculateDistance<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, Input, dCentroidsVectorData, dCentroidsDistance, dCentroidStarvation );
    // Kernel for finding the winning centroids
    sharedMem = (mStates+mStates)*sizeof(float);
    CalculateWinningCentroids<<<grid, threads, sharedMem>>>( mStates, dCentroidsDistance, dWinningCentroids );
    // Kernel for starvation updates
    UpdateStarvation<<<grid, threads>>>( mStates, mSTARVATION_COEFFICIENT, dWinningCentroids, dCentroidStarvation );
    // Kernel for updating winning centroids
    sharedMem = mInputDimensionlity*sizeof(float);
    UpdateWinningCentroids<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, mLearningRate, Input, dCentroidsVectorData, dWinningCentroids, dCentroidsDistance );
    // Kernel for calculating output
    sharedMem = (mStates+mStates)*sizeof(float);
    CalculatePOS<<<grid, threads, sharedMem>>>( mStates, dCentroidsDistance, dNodeOutput );

    this->WriteData(xml);
}

void DestinKernel::WriteData( stringstream& xml )
{
    hipMemcpy(mCentroidsDistance, dCentroidsDistance, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mCentroidStarvation, dCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mNodeOutput, dNodeOutput, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mWinningCentroids, dWinningCentroids, sizeOfNodes*sizeof(int), hipMemcpyDeviceToHost);

    xml << "<layer id=\"" << mID << "\">" << endl;
    for(int r=0;r<mRows;r++)
    {
        for(int c=0;c<mCols;c++)
        {
            int winningCentroid = mWinningCentroids[r*mCols+c];
            // winning counter finds place on the host might not be the best place to put this still
            // cause we are already writing here some output why create a special loop for it.
            mCentroidWinCounter[(c+r*mCols)*mStates+winningCentroid] += 1;
            xml << "<node id=\"" << r*mCols+c << "\" centroidWin=\"" << mWinningCentroids[r*mCols+c] << "\">" << endl;
            for(int s=0;s<mStates;s++)
            {
                xml << "<centroid id=\"" << s << "\" ";
                xml << "lastDistance=\"" << mCentroidsDistance[(c+r*mCols)*mStates+s] << "\" ";
                xml << "starvation=\"" << mCentroidStarvation[(c+r*mCols)*mStates+s] << "\" ";
                xml << "outPut=\"" << mNodeOutput[(c+r*mCols)*mStates+s]  << "\" ";
                xml << "winCount=\"" << mCentroidWinCounter[(c+r*mCols)*mStates+s]  << "\"";
                xml << "/>" << endl;
            }
            xml << "</node>" << endl;
        }
    }
    xml << "</layer>" << endl;
}
// ***********************
// DeSTIN inside CUDA Part
// ***********************
__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, float *CentroidStarvation)
{
    // This is how to declare a shared memory inside CUDA.
    extern __shared__ float shared[];
    float* input = (float*)&shared;
    float* distance = (float*)&input[InputDimensionlity];

    // We use many threads they need to know where they have to do there work.
    // tid (Thread ID) is the amount of threads inside a block its a fixed amount it can be changed by changing: AmountThreads.
    // Keep in mind that CUDA threads should be in steps of 32 (each warp takes 4 clock cycles where each cycle calculate 8 threads)
    int tid = threadIdx.x;
    // bid (Block ID) this keeps track in which  node we are working you can ask the grid the size of the blocks used in x or y and on a Fermi or higher even z
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    // make sure the input data is inside shared memory this we are going to compare the amount of centroids.
    while(tid < InputDimensionlity)
    {
        // Put input data for node inside shared memory
        input[tid] = InputData[tid + bid * InputDimensionlity];
        // A trick for when the dimension is bigger then the amount of threads
        tid += blockDim.x;
    }
    // all threads have to be here to be sure shared memory is filled with the input.
    __syncthreads();

    // calculation distance in massive thread style.
    // keep track of the centroid
    int centroid = 0;
    while (centroid<States)
    {
        // reset the tid
        tid = threadIdx.x;
        while(tid < InputDimensionlity)
        {
            // This temp will have for a short while the calculation of input - centroid for position tid (one cell of the vector)
            float temp = 0.0f;
            // distance to input = (input - centroid)*(input - centroid)
            // Small formula to get to the right working position: dimension*centroids*block+current centroid*dimension+thread
            temp = input[tid] - CentroidVectorData[InputDimensionlity*States*bid+centroid*InputDimensionlity+tid] ;
            distance[tid] = temp * temp;
            // A trick for when the dimension is bigger then the amount of threads
            tid += blockDim.x;
        }
        // all threads have to wait here so we know all distance have been calculated
        __syncthreads();

        // Cause DeSTIN don't work with numbers that are 2^? we have to check for odd numbers
        int dOld = InputDimensionlity;
        // bite wise divide by 2 (should be faster the /2)
        int d = InputDimensionlity >> 1;
        // a sum reduction, This is a common trick on CUDA to add shared memory instead of striding true memory
        // You have to use half the memory each step and each thread will add itself to with the other half.
        while (d != 0)
        {
            // reset the tid
            tid = threadIdx.x;
            dOld = dOld - d*2;
            while(tid < d)
            {
                // the adding calculation
                distance[tid] += distance[tid + d];

                // special case in case of odd number (As long as this doesn't happen too often it won't effect speed)
                if (dOld == 1 && tid == d-1)
                {
                    distance[tid] += distance[tid + d + 1];
                }
                tid += blockDim.x;
            }
            // Sync moment before starting with next iteration of reduction.
            __syncthreads();

            dOld = d;
            d >>= 1;
        }

        // Write distance to Node Data
        tid = threadIdx.x;
        if(tid == 0)
        {
            // square root on sum of the (input - centroid)*(input - centroid)
            // (Remember that you should copy the data from the device to the host and store it then)
            CentroidDist[centroid+bid*States] = (sqrt(distance[tid]))*CentroidStarvation[centroid+bid*States];
        }
        // go to next centroid inside the node (bid is taking care of the other node)
        centroid++;
    }
}

// To reduce the amount of work that one kernel is doing i have decided that splitting the work over more kernels should speed up the whole procces
__global__ void CalculateWinningCentroids( int States, float *CentroidDist, int *WinningCentroids )
{
    extern __shared__ float shared[];
    float* winner = (float*)&shared;
    float* winnerId = (float*)&winner[States];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    while(tid < States)
    {
        winnerId[tid] = tid;
        winner[tid] = CentroidDist[tid+bid*States];
        tid += blockDim.x;
    }
    __syncthreads();

    int dOld = States;
    int d = States >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            if(winner[tid] > winner[tid + d])
            {
                // Move winning centroid to the beginning
                winner[tid] = winner[tid + d];
                winnerId[tid] = winnerId[tid + d];
            }

            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                if(winner[tid] > winner[tid + d + 1])
                {
                    winner[tid] = winner[tid + d + 1];
                    winnerId[tid] = winnerId[tid + d + 1];
                }
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }
    // Write the winning centroid into there position
    tid = threadIdx.x;
    if(tid == 0)
    {
        WinningCentroids[bid] = winnerId[tid];
    }
}

// This is the updating starvation fast and quick to update all the nodes and reset the winning centroid
// According to DeSTIN paper: The winning centroid starvation gets reset while the others starve more
// Aldo this is the simple version of it it might be changed in the further cause this make the network also forget what it learn
// when it is looking at something else for a very long time (Short and Long term memory)
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation )
{
    // for tid and bid see CalculateDistance kernel.
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    while(tid < States)
    {
        // Let all centroid starve
        CentroidStarvation[tid+bid*States] = (1.0f-StarvationCoefficient)*CentroidStarvation[tid+bid*States];
        // Reset winning centroid
        CentroidStarvation[WinningCentroids[bid]+bid*States] = 1.0f;
        tid += blockDim.x;
    }
}

// Move the winning centroids closer to the observation
__global__ void UpdateWinningCentroids( int States, int InputDimensionlity, float LearningRate, float *InputData, float *CentroidVectorData, int *WinningCentroids, float *CentroidDist )
{
    extern __shared__ float newDistance[];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    int centroid = WinningCentroids[bid];
    float temp;
    float inputD;
    int pos;

    //this while block calculates the distance between the input vectors
    //and the centroid vectors
    while(tid < InputDimensionlity)
    {
        pos = InputDimensionlity*States*bid+centroid*InputDimensionlity+tid;
        temp = CentroidVectorData[pos];
        inputD = InputData[tid + bid * InputDimensionlity];
        temp = inputD - (temp * LearningRate);
        CentroidVectorData[pos] = temp;
        temp = (inputD - temp) * (inputD - temp);
        newDistance[tid] = temp;

        tid += blockDim.x;
        pos += blockDim.x;
    }
    __syncthreads();

    int dOld = InputDimensionlity;
    int d = InputDimensionlity >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            newDistance[tid] += newDistance[tid + d];
            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                newDistance[tid] += newDistance[tid + d + 1];
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }

    tid = threadIdx.x;
    if(tid == 0)
    {
        CentroidDist[centroid+bid*States] = sqrt(newDistance[0]);
    }
}

__global__ void CalculatePOS( int States, float *CentroidDist, float *Output )
{
    extern __shared__ float shared[];
    float* distance = (float*)&shared;
    float* tPOS = (float*)&distance[States];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    while(tid < States)
    {
        distance[tid] = CentroidDist[bid*States+tid];
        tPOS[tid] = (float)(1.0/(1e-9+(double)distance[tid]));
        tid += blockDim.x;
    }
    __syncthreads();

    int dOld = States;
    int d = States >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            tPOS[tid] += tPOS[tid + d];
            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                tPOS[tid] += tPOS[tid + d + 1];
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }

    tid = threadIdx.x;
    while(tid < States)
    {
        // This is the POS for all centroids (It looks like this is the input for the next layer also)
        // The output is missing the advice of higher layer
        Output[tid+bid*States] = (float)(1.0/(1e-9+(double)distance[tid]))/tPOS[0];
        tid += blockDim.x;
    }
}

__global__ void UpdateCountingTables(int mStates, int parentStates, int * dCountingTables,
		int * dParentsAdvice, int * dOldWinningStates, int * dNewWinningNodes, int * dSumTables){

	 int bid = blockIdx.x + blockIdx.y * gridDim.x;

	 int s2 = mStates * mStates;

	 //make sure Im consistent with old states across the top and new states down the side for the table.
	 int i = bid * parentsStates * s2
			 + dParentsAdvice[bid] * s2
			 + dOldWinningState[bid] * mStates
			 + dNewWinningState[bid];

	 dCountingTables[i]++;

	 i = bid * parentStates * mStates +
			 + dParentsAdvice[bid] * mStates
			 + dOldWinningStates[bid];

	 dSumTables[i]++;

}

/*

	dim3 grid( mCols, mRows);
	dim3 threads( states, states);

	UpdateBeliefs<< grid, threads >>();


*/

//TODO: make a check to see if it has enough shared memory
__global__ void UpdateBeliefs( int mStates, float *CentroidDist,
		float *dPOS, float * dNewBeliefs, float * dOldBeliefs, float * dCountingTables,
		int * dParentsAdvice, int parentsStates, int * dSumTables){

	int bid = blockIdx.x + blockIdx.y * gridDim.x; //corresponds to the node

	int advice = dParentsAdvice[bid];

	const int s2 = mStates * mStates;

	// Variable cts (counting table start) is the first element of the correct PSSA counting table
	// based on the node and advice state. PSSA means P(s'|s,c)*b(s) where c = advice, meaning
	// probability of transitioning to state s' given the current state s and the parents node's advice (or state) c.
	// Each node has a seperate counting table for each possible parent advice state
	// of size N x N, where N is the number of centroids (states) of the child node.
	// The number of counting tables per node equal to the number of parent states.
	// Each time a node transitions from s to s' given advice c, the counting table for advice c
	// has the value of the element at row s and column s' incremented by 1. Then, to get the
	// probability, that value is divided by the corresponding value in the SumsTables.
	// There is one sum table per counting table, which has one element per column of the
	// matching counting table which sums up the elements of the column.
	const int cts = bid * parentsStates * s2 + advice * s2;


	__shared__ float cache[s2]; // the cache saves each P(s'|s,c)*b(s) for s = 0 to mStates - 1.

	//variable sp is read as "s prime" as in b'(s') which is the left side of the belief update equation.
	for(int sp = threadIdx.y; sp < mStates ; sp += blockDim.y) {
		int ctr = cts + sp * mStates; // ctr (counting table row) is the first element of the sp'th row of the counting table
		//s = current state
		for(int s = threadIdx.x ;  s < mStates ; s += blockDim.x ) {
			int i = ctr + s;
			//TODO: is it guaranteed that dSumTables will not be 0?
			float prob = dCountingTables[i] / dSumTables[bid * parentStates * mStates	+ advice * mStates + s];
			cache[ sp * mStates + s] = dOldBeliefs[bid * mStates + s] * prob; // this is the P(s'|s,c)*b(s) calculation.

		}
	}
	__syncthreads();

	__shared__ float sums[mStates]; //hold sum of P(s'|s,c)*b(s)
	//should probably do a reduction trick here
	if(threadIdx.x == 0){
		for(int sp = threadIdx.y; sp < mStates ; sp += blockDim.y){
			sums[sp] = 0;
			for(int s = 0 ; s < mStates ; s++){
				sums[sp] += cache[ sp * mStates +  s ];
			}
		}
	}
	__syncthreads();

	for(int sp = threadIdx.y ; sp < mStates ; sp += blockDim.y){
		int i = bid * mStates + sp;
		dNewBeliefs[i] = dPOS[i] * sums[sp];
	}

}


