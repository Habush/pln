#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>

// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int AmountThreads = 128;


using namespace std;

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, float *CentroidStarvation);
__global__ void CalculateWinningCentroids( int States, float *CentroidDist, int *WinningCentroids );
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation );
__global__ void UpdateWinningCentroids( int States, int InputDimensionlity, float LearningRate, float *InputData, float *CentroidVectorData, int *WinningCentroids, float *CentroidDist );
__global__ void CalculatePOS( int States, float *CentroidDist, float *Output );
__global__ void UpdateBeliefs( const int states, float *dPOS, float * dNewBeliefs, float * dOldBeliefs, int * dCountingTables, int * dParentsAdvice, int parentStates, int * dSumTables, int * dOutputAdvice);
__device__ void find_max(const int states,float * winner, int * winnerId );
__device__ void updateCountingTables(int mStates, int parentStates, int * dCountingTables, int advice, int previousWinningBelief, int newWinningBelief, int * dSumTables, int bid);
__global__ void initializeMemory( const int states, const int parentStates, float * dBeliefs, int * dOutputAdvice, int * dCountingTables, int * dSumTables );

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mParentStates=0;
	mInputDimensionlity=0;
    mLearningRate = 0;
    mSTARVATION_COEFFICIENT = 0;
	hipGetDeviceCount(&mDevices);
	cout << "Kernel created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    hipFree( dCentroidsVectorData );
    hipFree( dPOS );
    hipFree( dCentroidsDistance );
    hipFree( dCentroidStarvation );
    hipFree( dWinningCentroids );
    hipFree( dBeliefs );
    hipFree( dOutputAdvice );
    hipFree( dCountingTables );
    hipFree( dSumTables );

    free ( mCentroidsDistance );
    free ( mCentroidStarvation );
    free ( mWinningCentroids );
    free ( mPOS );
    free(mCentroidWinCounter);
    cout << "Kernel destroyed" << endl;
}

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int ParentStates, int InputDimensionlity, float FixedLeaningRate, hiprandGenerator_t gen)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mParentStates = ParentStates;
    mInputDimensionlity = InputDimensionlity;
    mLearningRate = FixedLeaningRate;

    mSTARVATION_COEFFICIENT = 1.0/((float)InputDimensionlity*(float)InputDimensionlity);
    if ( mSTARVATION_COEFFICIENT < 1.0/512.0 )
    {
        mSTARVATION_COEFFICIENT=1.0/512.0;
    }

    // Define the data sizes
    // Size of de nodes is rows times columns
    sizeOfNodes = mRows*mCols;
    // Size of the data of nodes is rows times columns times centroids
    sizeOfNodeData = sizeOfNodes*mStates;

    // Size of the layer with all vectors is rows times columns times centroids times the input (also observation) vector length.
    sizeOfLayerData = sizeOfNodeData*mInputDimensionlity;
    // Keep track of which centroid won
    mCentroidWinCounter = new int[sizeOfNodeData];
    for(int c=0;c<sizeOfNodeData;c++)
    {
        mCentroidWinCounter[c] = 0;
    }

    //TODO: put in error checking incase the cudaMallocs fail in case of not enough memory on device
    // Array full with all the winning centroids of each node
    mWinningCentroids = new int[sizeOfNodes];
    hipMalloc( (void**)&dWinningCentroids, sizeOfNodes*sizeof(int) );

    // Node data contains the distance to the observation of all centroids (It's is empty the first run)
    mCentroidsDistance = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidsDistance, sizeOfNodeData*sizeof(float) );

    // Starvation data for all centroids
    mCentroidStarvation = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidStarvation, sizeOfNodeData*sizeof(float) );
    for(int i=0;i<sizeOfNodeData;i++)
    {
        mCentroidStarvation[i]=1.0f;
    }
    // Copy the data from host to device
    hipMemcpy(dCentroidStarvation, mCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyHostToDevice);

    //TODO: make sure this POS is being fed to the correct place, and if it needs to go back to the host
    //POS - P(o|s') of update equation
    mPOS = new float[sizeOfNodeData];
    hipMalloc( (void**)&dPOS, sizeOfNodeData*sizeof(float) );

    hipMalloc( (void**)&dCentroidsVectorData, sizeOfLayerData*sizeof(float) );

    // This is to fill the dLayerData with all random numbers between 0.0 and 1.0
    hiprandGenerateUniform( gen, dCentroidsVectorData, sizeOfLayerData );

    //Node belief output, fed as input to parent nodes
    hipMalloc((void**)&dBeliefs, sizeOfNodeData * sizeof(float));

    //Node advice for fed to child nodes
    hipMalloc((void**)&dOutputAdvice, sizeOfNodes * sizeof(int));

    //Used in P(s'|s,a) calculations, counts when node transitions from s to s' when parent advice = a
    hipMalloc((void**)&dCountingTables, mRows * mCols * mParentStates * mStates * mStates * sizeof(int));

    //Used in P(s'|s,a) (aka PSSA) calculations, holds the sum of the counting table columns
    hipMalloc((void**)&dSumTables, mRows * mCols * mParentStates * mStates * sizeof(int));

    dim3 grid(mCols, mRows); //grid of nodes (aka blocks)
    dim3 threads(AmountThreads);//threads per node

    //initialize memory to uniform distribution
    initializeMemory<<<grid, threads>>>(mStates, mParentStates, dBeliefs, dOutputAdvice, dCountingTables, dSumTables);
}

__global__ void initializeMemory( const int states, const int parentStates, float * dBeliefs, int * dOutputAdvice, int * dCountingTables, int * dSumTables ){
	int bid = blockIdx.x + blockIdx.y * gridDim.x;
	int tid = threadIdx.x;
	//dBeliefs[bid]
	int threads = blockDim.x ;

	const float uniform_c = 1.0 / (float)states;
	for(int t = tid ; t < states ; t+=threads ){
		dBeliefs[bid*states+t] = uniform_c;
	}

	if(tid==0){
		dOutputAdvice[bid] = 0;
	}

	//Each node has N=parentStates counting tables, each is size states x states
	for(int t = tid ; t < parentStates * states * states ; t+=threads){
		dCountingTables[bid * parentStates * states * states + t] = 1;
	}
	//holds the sums of the columns of the counting tables
	for(int t = tid ; t < parentStates * states ; t+=threads){
		dSumTables[bid*parentStates * states + t ] = states;
	}
}

void DestinKernel::DoDestin( float *Input, stringstream& xml )
{
    // Threads is the amount of thread inside each block
    dim3 threads( AmountThreads );
    // Grid is the amount of blocks inside a grid.
    dim3 grid( mCols, mRows );
    // Cause of the use of dynamic shared memory you have to tell the kernel how much shared memory space you need for each block.
    int sharedMem;
    // The launch of the kernels itself with centroids(states), dimension, input data and the Data of the layer itself
    // Calculating the distance of the centroids to an observation
    sharedMem = (mInputDimensionlity+mInputDimensionlity)*sizeof(float);
    CalculateDistance<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, Input, dCentroidsVectorData, dCentroidsDistance, dCentroidStarvation );
    // Kernel for finding the winning centroids
    sharedMem = (mStates+mStates)*sizeof(float);
    CalculateWinningCentroids<<<grid, threads, sharedMem>>>( mStates, dCentroidsDistance, dWinningCentroids );
    // Kernel for starvation updates
    UpdateStarvation<<<grid, threads>>>( mStates, mSTARVATION_COEFFICIENT, dWinningCentroids, dCentroidStarvation );
    // Kernel for updating winning centroids
    sharedMem = mInputDimensionlity*sizeof(float);
    UpdateWinningCentroids<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, mLearningRate, Input, dCentroidsVectorData, dWinningCentroids, dCentroidsDistance );
    // Kernel for calculating output
    sharedMem = (mStates+mStates)*sizeof(float);
    CalculatePOS<<<grid, threads, sharedMem>>>( mStates, dCentroidsDistance, dPOS );

    //TODO: rename dNewBeliefs and dOldBeliefs to just dBeliefs, get rid of one.
    //TODO: make sure dParentsAdvice is correct
    //TODO: set parentsStates in create function
    int n = mStates > 16 ? 16 : mStates ;
    dim3 threads_plane( n, n ); 
    //total threads should be less than 512 per block, hardware limit so states needs to be less<=22
    //Chose 16 because seems like it would play better than 22... but not sure.
    //TODO: this should probably be a multiple of states instead to avoid wasting threads
    //TODO: update dBeliefs properly
    //TODO: might make sense to break up UpdateBeliefs because alot is done with just a single row of threads so
    //lots of the threads are wasted, not sure if this would outweight the overhead of a sperate kernel launchss
    sharedMem = (mStates * mStates + mStates) * sizeof(float);

    UpdateBeliefs<<<grid, threads_plane, sharedMem >>>(mStates, dPOS, dBeliefs, dBeliefs, dCountingTables, dParentInputAdvice, mParentStates, dSumTables, dOutputAdvice) ;
    

    this->WriteData(xml);
}

void DestinKernel::WriteData( stringstream& xml )
{
    hipMemcpy(mCentroidsDistance, dCentroidsDistance, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mCentroidStarvation, dCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mPOS, dPOS, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mWinningCentroids, dWinningCentroids, sizeOfNodes*sizeof(int), hipMemcpyDeviceToHost);

    xml << "<layer id=\"" << mID << "\">" << endl;
    for(int r=0;r<mRows;r++)
    {
        for(int c=0;c<mCols;c++)
        {
            int winningCentroid = mWinningCentroids[r*mCols+c];
            // winning counter finds place on the host might not be the best place to put this still
            // cause we are already writing here some output why create a special loop for it.
            mCentroidWinCounter[(c+r*mCols)*mStates+winningCentroid] += 1;
            xml << "<node id=\"" << r*mCols+c << "\" centroidWin=\"" << mWinningCentroids[r*mCols+c] << "\">" << endl;
            for(int s=0;s<mStates;s++)
            {
                xml << "<centroid id=\"" << s << "\" ";
                xml << "lastDistance=\"" << mCentroidsDistance[(c+r*mCols)*mStates+s] << "\" ";
                xml << "starvation=\"" << mCentroidStarvation[(c+r*mCols)*mStates+s] << "\" ";
                xml << "POS=\"" << mPOS[(c+r*mCols)*mStates+s]  << "\" ";
                xml << "winCount=\"" << mCentroidWinCounter[(c+r*mCols)*mStates+s]  << "\"";
                xml << "/>" << endl;
            }
            xml << "</node>" << endl;
        }
    }
    xml << "</layer>" << endl;
}
// ***********************
// DeSTIN inside CUDA Part
// ***********************
__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, float *CentroidStarvation)
{
    // This is how to declare a shared memory inside CUDA.
    extern __shared__ float shared[];
    float* input = (float*)&shared;
    float* distance = (float*)&input[InputDimensionlity];

    // We use many threads they need to know where they have to do there work.
    // tid (Thread ID) is the amount of threads inside a block its a fixed amount it can be changed by changing: AmountThreads.
    // Keep in mind that CUDA threads should be in steps of 32 (each warp takes 4 clock cycles where each cycle calculate 8 threads)
    int tid = threadIdx.x;
    // bid (Block ID) this keeps track in which  node we are working you can ask the grid the size of the blocks used in x or y and on a Fermi or higher even z
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    // make sure the input data is inside shared memory this we are going to compare the amount of centroids.
    while(tid < InputDimensionlity)
    {
        // Put input data for node inside shared memory
        input[tid] = InputData[tid + bid * InputDimensionlity];
        // A trick for when the dimension is bigger then the amount of threads
        tid += blockDim.x;
    }
    // all threads have to be here to be sure shared memory is filled with the input.
    __syncthreads();

    // calculation distance in massive thread style.
    // keep track of the centroid
    int centroid = 0;
    while (centroid<States)
    {
        // reset the tid
        tid = threadIdx.x;
        while(tid < InputDimensionlity)
        {
            // This temp will have for a short while the calculation of input - centroid for position tid (one cell of the vector)
            float temp = 0.0f;
            // distance to input = (input - centroid)*(input - centroid)
            // Small formula to get to the right working position: dimension*centroids*block+current centroid*dimension+thread
            temp = input[tid] - CentroidVectorData[InputDimensionlity*States*bid+centroid*InputDimensionlity+tid] ;
            distance[tid] = temp * temp;
            // A trick for when the dimension is bigger then the amount of threads
            tid += blockDim.x;
        }
        // all threads have to wait here so we know all distance have been calculated
        __syncthreads();

        // Cause DeSTIN don't work with numbers that are 2^? we have to check for odd numbers
        int dOld = InputDimensionlity;
        // bite wise divide by 2 (should be faster the /2)
        int d = InputDimensionlity >> 1;
        // a sum reduction, This is a common trick on CUDA to add shared memory instead of striding true memory
        // You have to use half the memory each step and each thread will add itself to with the other half.
        while (d != 0)
        {
            // reset the tid
            tid = threadIdx.x;
            dOld = dOld - d*2;
            while(tid < d)
            {
                // the adding calculation
                distance[tid] += distance[tid + d];

                // special case in case of odd number (As long as this doesn't happen too often it won't effect speed)
                if (dOld == 1 && tid == d-1)
                {
                    distance[tid] += distance[tid + d + 1];
                }
                tid += blockDim.x;
            }
            // Sync moment before starting with next iteration of reduction.
            __syncthreads();

            dOld = d;
            d >>= 1;
        }

        // Write distance to Node Data
        tid = threadIdx.x;
        if(tid == 0)
        {
            // square root on sum of the (input - centroid)*(input - centroid)
            // (Remember that you should copy the data from the device to the host and store it then)
            CentroidDist[centroid+bid*States] = (sqrt(distance[tid]))*CentroidStarvation[centroid+bid*States];
        }
        // go to next centroid inside the node (bid is taking care of the other node)
        centroid++;
    }
}

	// To reduce the amount of work that one kernel is doing i have decided that splitting the work over more kernels should speed up the whole procces
   //TODO: make sure this can work with __shared__ as is
__global__ void CalculateWinningCentroids( int States, float *CentroidDist, int *WinningCentroids )
{
    extern __shared__ float shared[];
    float* winner = (float*)&shared;
    float* winnerId = (float*)&winner[States];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    while(tid < States)
    {
        winnerId[tid] = tid;
        winner[tid] = CentroidDist[tid+bid*States];
        tid += blockDim.x;
    }
    __syncthreads();

    int dOld = States;
    int d = States >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            if(winner[tid] > winner[tid + d])
            {
                // Move winning centroid to the beginning
                winner[tid] = winner[tid + d];
                winnerId[tid] = winnerId[tid + d];
            }

            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                if(winner[tid] > winner[tid + d + 1])
                {
                    winner[tid] = winner[tid + d + 1];
                    winnerId[tid] = winnerId[tid + d + 1];
                }
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }
    // Write the winning centroid into there position
    tid = threadIdx.x;
    if(tid == 0)
    {
        WinningCentroids[bid] = winnerId[tid];
    }
}

// This is the updating starvation fast and quick to update all the nodes and reset the winning centroid
// According to DeSTIN paper: The winning centroid starvation gets reset while the others starve more
// Aldo this is the simple version of it it might be changed in the further cause this make the network also forget what it learn
// when it is looking at something else for a very long time (Short and Long term memory)
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation )
{
    // for tid and bid see CalculateDistance kernel.
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    while(tid < States)
    {
        // Let all centroid starve
        CentroidStarvation[tid+bid*States] = (1.0f-StarvationCoefficient)*CentroidStarvation[tid+bid*States];
        // Reset winning centroid
        CentroidStarvation[WinningCentroids[bid]+bid*States] = 1.0f;
        tid += blockDim.x;
    }
}

// Move the winning centroids closer to the observation
__global__ void UpdateWinningCentroids( int States, int InputDimensionlity, float LearningRate, float *InputData, float *CentroidVectorData, int *WinningCentroids, float *CentroidDist )
{
    extern __shared__ float newDistance[];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    int centroid = WinningCentroids[bid];
    float temp;
    float inputD;
    int pos;

    //this while block calculates the distance between the input vectors
    //and the centroid vectors
    while(tid < InputDimensionlity)
    {
        pos = InputDimensionlity*States*bid+centroid*InputDimensionlity+tid;
        temp = CentroidVectorData[pos];
        inputD = InputData[tid + bid * InputDimensionlity];
        temp = inputD - (temp * LearningRate);
        CentroidVectorData[pos] = temp;
        temp = (inputD - temp) * (inputD - temp);
        newDistance[tid] = temp;

        tid += blockDim.x;
        pos += blockDim.x;
    }
    __syncthreads();

    int dOld = InputDimensionlity;
    int d = InputDimensionlity >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            newDistance[tid] += newDistance[tid + d];
            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                newDistance[tid] += newDistance[tid + d + 1];
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }

    tid = threadIdx.x;
    if(tid == 0)
    {
        CentroidDist[centroid+bid*States] = sqrt(newDistance[0]);
    }
}

__global__ void CalculatePOS( int States, float *CentroidDist, float *POSOutput )
{
    extern __shared__ float shared[];
    float* distance = (float*)&shared;
    float* tPOS = (float*)&distance[States];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    while(tid < States)
    {
        distance[tid] = CentroidDist[bid*States+tid];
        tPOS[tid] = (float)(1.0/(1e-9+(double)distance[tid]));
        tid += blockDim.x;
    }
    __syncthreads();

    int dOld = States;
    int d = States >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            tPOS[tid] += tPOS[tid + d];
            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                tPOS[tid] += tPOS[tid + d + 1];
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }

    tid = threadIdx.x;
    while(tid < States)
    {
        // This is the POS for all centroids (It looks like this is the input for the next layer also)
        // The output is missing the advice of higher layer
    	POSOutput[tid+bid*States] = (float)(1.0/(1e-9+(double)distance[tid]))/tPOS[0];
        tid += blockDim.x;
    }
}



/*

	dim3 grid( mCols, mRows);

	int n = states > 22 ? 22 : states;
	dim3 threads( n, n ); //total threads should be less than 512 per block, hardware limit so states needs to be less<=22
	//TODO: this should probably be a multiple of states instead to avoid wasting threads

	UpdateBeliefs<< grid, threads >>();


*/

//TODO: make a check to see if it has enough shared memory
//TODO: make sure that I'm useing __shared__ properly and if im doing it dynamically will it work properly
//TODO: still need to normalize as in the denominator of the belief update equation.

/**
 * UpdateBeliefs - Performs the P(s'|s,a)*b(s) calculations of the DeSTIN belief update rule.
 *
 * PSSA means P(s'|s,a)*b(s) where a = advice, meaning
 * probability of transitioning to state s' given the current state s and the parents node's advice (or state) a.
 * Each node has a seperate counting table for each possible parent advice state
 * of size N x N, where N is the number of centroids (states) of the child node.
 * The number of counting tables per node equal to the number of parent states.
 * Each time a node transitions from s to s' given advice a, the counting table for advice a
 * has the value of the element at row s and column s' incremented by 1. Then, to get the
 * probability, that value is divided by the corresponding value in the SumsTables.
 * There is one sum table per counting table, which has one element per column of the
 * matching counting table which sums up the elements of the column
 *
 * states - number of node centroids
 * dPOS - P(o|s') calculated from CalculatePOS kernel
 * dNewBeliefs - b'(s') - updated beliefs. The node output, fed to parent nodes as input
 * dOldBeliefs - b(s) - beliefs how they were before calling this kernel, currently dOldBeliefs points to same memory location as dNewBeliefs
 * dCountingTables - keeps track of the P(s'|s,a) table along with the dSumTables
 * dParentsInputAdvice - input advice from the parent node. The 'a' of P(s'|s,a). NULL if this is the top layer, no parent layer.
 * parentStates - number of centroids of the parent node. Zero if this is the top layer.
 * dSumTables - vector of the sum of the columns of the dCountingTables
 * dOutputAdvice - this node's advice to be fed to its children nodes
 */
__global__ void UpdateBeliefs( const int states, float *dPOS, float * dNewBeliefs, float * dOldBeliefs, int * dCountingTables,
		int * dParentsInputAdvice, int parentStates, int * dSumTables, int * dOutputAdvice){

	int bid = blockIdx.x + blockIdx.y * gridDim.x; //corresponds to the node

	//TODO: enforce square layers or update child to parent mapping code to handle non square layers
	//Points the 4 children nodes to the right parent for advice
	//Be careful of integer division if trying to simplify this.
	//TODO: simplify this
	//TODO: could make one thread pull from global memory into shared memory, instead of all threads pulling
	int parent_node_id = blockIdx.x / 2 + (blockIdx.y /2 ) * (gridDim.x / 2) ;
	const int advice = dParentsInputAdvice==NULL ? 0 : dParentsInputAdvice[parent_node_id];

	const int s2 = states * states;

	// Variable cts (counting table start) is the first element (at 0,0) of the correct PSSA counting table
	// based on the node and advice state.
	const int cts = bid * parentStates * s2 + advice * s2;


	extern __shared__ float cache[]; // the cache saves each P(s'|s,c)*b(s) for all s' and s for the given advice c. The size  is states x states ( plus another states tacked on the end later, see kernel launch params)

	//variable sp is read as "s prime" as in b'(s') which is the left side of the belief update equation.
	for(int sp = threadIdx.y; sp < states ; sp += blockDim.y) {
		int ctr = cts + sp * states; // ctr (counting table row) is the first element of the sp'th row of the counting table
		//s = current state
		for(int s = threadIdx.x ;  s < states ; s += blockDim.x ) {
			int i = ctr + s;
			//TODO: i might be performing this multiplication in the wrong order
			float prob = (float)dCountingTables[i] / (float) dSumTables[bid * parentStates * states + advice * states + s];
			//TODO: should probably save the dOldBeliefs vector to a shared memory variable first
			// to prevent having to pull it from global memory N=states times
			cache[ sp * states + s] = dOldBeliefs[bid * states + s] * prob; // this is the P(s'|s,c)*b(s) calculation.
		}
	}
	__syncthreads();



	//this part performs a reduction on the sums of the P(s'|s,c)*b(s) rows
	//of the cache table, storing the sums in the first column of the table.
	int dOld = states;
	for (int d = states >> 1;  d != 0; d >>= 1) { 				
		dOld -= d*2;	
		for(int sp = threadIdx.y; sp < states ; sp += blockDim.y){
			for(int s = threadIdx.x; s < d ; s += blockDim.x){
				int i = sp * states + s;
				cache[i] +=  cache[i + d];
				//trick for if cache has odd length
				if(dOld == 1 && s == d - 1){
					cache[i] += cache[i + d + 1];
				}
			}
		}
		__syncthreads(); //TODO: is this the correct place for the sync?
		dOld = d;
	}

	float * pssc_b_vector = (float *)&cache[s2]; //length is states, start this vector right after the end of cache array

	//multiply the two parts of the belief update equation numerator together, Pr(o|s') by Sum[ Pr(s'|s,c)*b(S) ]
	//The cache[sp * states] is the Pr(s'|s,c)*b(S)  vector

     //we launched with a 2d block of threads now only dealing with 1d arrays, so convert this back to 1d so we waste fewer threads
	int sp_start  = threadIdx.y * blockDim.x + threadIdx.x; 
    int n_threads = blockDim.x * blockDim.y;

    //transform it from a column into a row
	for(int sp = sp_start ; sp < states ; sp += n_threads ){
		//dNewBeliefs[i] = dPOS[i] * cache[sp * states];
		pssc_b_vector[sp] = cache[sp * states] *= dPOS[bid * states + sp];
	}
	__syncthreads(); //might not need this here


	//find the sum of the pssc_b_vector so it can be normalized   
	dOld = states;
	int s_start = sp_start;   
	for (int d = states >> 1; d!=0 ; dOld = d, d>>=1 ){
		dOld -= d*2;
		for(int s = s_start; s < d; s +=  n_threads ){
			pssc_b_vector[s] += pssc_b_vector[s + d ];
			if(dOld == 1 && s ==  d - 1){
				pssc_b_vector[s] += pssc_b_vector[s + d + 1];
			}
		}
		__syncthreads();	
	}

	float sum = pssc_b_vector[0];


	//pssc_b_vector will now contain b'(s') after normalizing the Pr(s'|s,c)*b(S) vector
	for(int sp = sp_start; sp < states ; sp += n_threads ){
		dNewBeliefs[ bid * states + sp] =  pssc_b_vector[sp] = cache[sp * states] /= sum;
	}
	
	int * max_index = (int *)cache; //max_index size = #states. Overwrite first row of cache shared memory to save winning index.

	//find max belief, store corresponding index in max_index[0]
	find_max(states, pssc_b_vector, max_index);

	//set max belief state as advice for child nodes
	if(sp_start == 0){//only one thread does this to save memory bandwidth
		int old_winning_belief = dOutputAdvice[bid];
		//new winning belief
		dOutputAdvice[bid] = max_index[0];
		updateCountingTables(states, parentStates, dCountingTables, advice, old_winning_belief, max_index[0], dSumTables, bid);
	}

}

__device__ void updateCountingTables(int mStates, int parentStates, int * dCountingTables,
			int advice, int previousWinningBelief, int newWinningBelief, int * dSumTables, int bid){
	 int s2 = mStates * mStates;
	 //make sure Im consistent with old states across the top and new states down the side for the table.
	 int i = bid * parentStates * s2 //node
			 + advice * s2 //advice table for node
			 + newWinningBelief * mStates //row of table
			 + previousWinningBelief;	//col of table

	 dCountingTables[i]++;

	 //dSumTables, collection of 1 dimensional vectors. Each node has the same number of them as its counting tables or one for each parent state.
	 //One sum vector has length equal to the number of the node's centroids or states. Each element is the sum of the corresponding column
	 //of the dCountingTable
	 i = bid * parentStates * mStates // node index
		 + advice * mStates //advice index
		 + previousWinningBelief; //element of sum vector
	 dSumTables[i]++;

}

__device__ void find_max(const int states,float * winner, int * winnerId ){

    int tid;
    for(tid = threadIdx.x; tid < states ; tid += blockDim.x ){
        winnerId[tid] = tid;
    }
    __syncthreads();
    
    for(int dOld = states, d = states >> 1; d != 0 ; dOld = d, d >>= 1 ){
        for(tid = threadIdx.x, dOld -= d*2 ; tid < d ; tid += blockDim.x){
            int tidd= tid + d;	
            if(winner[tid] > winner[tidd]){
                // Move winning centroid to the beginning
                winner[tid] = winner[tidd];
                winnerId[tid] = winnerId[tidd];
            }
            if (dOld == 1 && tid == d-1){
                // special case of odd numbers
                if(winner[tid] > winner[tidd + 1]){
                    winner[tid] = winner[tidd + 1];
                    winnerId[tid] = winnerId[tidd + 1];
                }
            }
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();
    }
}

